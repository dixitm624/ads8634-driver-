#include <hip/hip_runtime.h>

#include <iostream>
#include <ctime>
#include <chrono>

float measure_kernel_execution_time(hipEvent_t start, hipEvent_t stop)
{
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    return milliseconds;
}

__global__ void haze_removal_kernel(const unsigned char *src, int rows, int cols, int channels, int A, const unsigned char *tx, unsigned char *result_img)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;

    if (idx < cols && idy < rows)
    {
        int index = (idy * cols + idx) * channels;
        float max_t = static_cast<float>(tx[idy * cols + idx]) / 255.0f;
        if (max_t < 0.5f)
        {
            max_t = 0.5f;
        }

        for (int c = 0; c < channels; ++c)
        {
            int value_num = src[index + c];
            result_img[index + c] = static_cast<unsigned char>((value_num - A) / max_t + A);
        }


    }
}

__global__ void calculate_tx_kernel(int rows, int cols, int channels, float A, unsigned char *dark_channel_data, unsigned char *tx_data)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;

    if (idx < cols && idy < rows)
    {

        float dark_channel_val = static_cast<float>(dark_channel_data[idy * cols + idx]) / A;

        tx_data[idy * cols + idx] = static_cast<unsigned char>(255.0f * (1.0f - (0.95f * dark_channel_val)));
    }
}

__global__ void sortIndicesByDarkChannel(const unsigned char *darkChannel, int *sortedIndices, int numRows, int numCols)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int idx = row * numCols + col;

    if (row < numRows && col < numCols)
    {
        sortedIndices[idx] = idx;
    }
    __syncthreads();

    if (row < numRows && col < numCols)
    {
        int temp = sortedIndices[idx];
        int j = idx - 1;
        while (j >= 0 && darkChannel[temp] > darkChannel[sortedIndices[j]])
        {
            sortedIndices[j + 1] = sortedIndices[j];
            j--;
        }
        sortedIndices[j + 1] = temp;
    }
}

__global__ void findMaxValue(const unsigned char *imageData, const int *sortedIndices, int numRows, int numCols, float *maxVal)
{
    // Calculate the number of elements for 1%
    int numElements = numRows * numCols;
    int onePercent = numElements;

    // Each thread handles one element
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    // Initialize shared memory to store intermediate maximum values
    __shared__ float s_maxVal[256];
    s_maxVal[threadIdx.x] = 0.0f;

    // Fetch the indices from sortedIndices and find their corresponding pixels in the RGB image
    if (idx < onePercent)
    {
        int sortedIndex = sortedIndices[idx];
        unsigned char r = imageData[sortedIndex * 3];
        unsigned char g = imageData[sortedIndex * 3 + 1];
        unsigned char b = imageData[sortedIndex * 3 + 2];

        // Calculate the maximum value for this thread
        float threadMax = fmaxf(fmaxf(r, g), b);

        // Store the maximum value in shared memory
        s_maxVal[threadIdx.x] = threadMax;
    }

    // Synchronize threads to ensure all values are stored in shared memory
    __syncthreads();

    // Perform parallel reduction to find the maximum value across all threads
    #pragma unroll
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1)
    {
        if (threadIdx.x < stride)
        {
            s_maxVal[threadIdx.x] = fmaxf(s_maxVal[threadIdx.x], s_maxVal[threadIdx.x + stride]);
        }
        __syncthreads();
    }

    // Store the final maximum value to global memory
    if (threadIdx.x == 0)
    {
        atomicMax(reinterpret_cast<unsigned int *>(maxVal), *reinterpret_cast<unsigned int *>(&s_maxVal[0]));
    }
}

__global__ void min_channel_kernel(unsigned char *src_data, unsigned char *min_mat_data, int rows, int cols, int channels)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    int index = idy * cols + idx;

    if (idx < cols && idy < rows)
    {
        int min_val = 255;
        #pragma unroll
        for (int c = 0; c < channels; ++c)
        {
            int val = src_data[index * channels + c];
            min_val = min(min_val, val);
        }
        min_mat_data[index] = min_val;
    }
}

__global__ void min_filter_kernel(unsigned char *min_mat_expansion_data, unsigned char *dark_channel_mat_data, int rows, int cols, int border)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    int m = idy + border;
    int n = idx + border;

    if (m < rows - border && n < cols - border)
    {
        int min_num = 255;
        #pragma unroll
        for (int i = -border; i <= border; ++i)
        {
            #pragma unroll
            for (int j = -border; j <= border; ++j)
            {
                int val_roi = min_mat_expansion_data[(m + i) * cols + n + j];
                min_num = min(min_num, val_roi);
            }
        }
        dark_channel_mat_data[(m - border) * (cols - 2 * border) + (n - border)] = min_num;
    }
}

__global__ void padImage(unsigned char *input, unsigned char *output, int rows, int cols, int borderSize)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    // Padding top border
    if (i < cols && j < borderSize)
    {
        int idx_out = i + borderSize + (j * (cols + 2 * borderSize) + borderSize);
        int idx_in = i + j * cols;
        output[idx_out] = input[idx_in];
    }

    // Padding bottom border
    if (i < cols && j >= rows - borderSize)
    {
        int idx_out = i + borderSize + ((j + borderSize * 2) * (cols + 2 * borderSize) + borderSize);
        int idx_in = i + (rows - borderSize - 1) * cols + (j - (rows - borderSize)) * cols;
        output[idx_out] = input[idx_in];
    }

    // Padding left border
    if (i < borderSize && j < rows)
    {
        int idx_out = i + (j * (cols + 2 * borderSize));
        int idx_in = j * cols;
        output[idx_out] = input[idx_in];
    }

    // Padding right border
    if (i >= cols && i < cols + borderSize && j < rows)
    {
        int idx_out = (i + borderSize) + (j * (cols + 2 * borderSize)) + borderSize;
        int idx_in = (i - cols + 1) + j * cols;
        output[idx_out] = input[idx_in];
    }

    if (i < cols && j < rows)
    {
        int idx_in = i + j * cols;
        int idx_out = (i + borderSize) + (j + borderSize) * (cols + 2 * borderSize);
        output[idx_out] = input[idx_in];
    }
}



void print_data(unsigned char *data, int rows, int cols, int channels)
{
    std::cout << "===============================start=================================" << std::endl;
    for (int i = 0; i < rows - 400; ++i)
    {
        for (int j = 0; j < cols - 400; ++j)
        {
            for (int c = 0; c < channels; ++c)
            {
                std::cout << static_cast<int>(data[(i * cols + j) * channels + c]) << " ";
            }
            std::cout << "| ";
        }
        std::cout << std::endl;
    }

    std::cout << "===============================end==================================" << std::endl;
}








unsigned char* Dehaze_CUDA(unsigned char *src_ptr, int rows, int cols, int channels, int border)
{


    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Record the start event
    hipEventRecord(start);
    //Varible Decleration
    unsigned char *src_data;
    unsigned char *min_mat_data;
    unsigned char *dark_channel_mat_data;
    unsigned char *min_mat_expansion_data;
    int* sortedIndices;
    float* A;
    unsigned char* dev_tx_data = 0;
    unsigned char* result;
    unsigned char * result_img;


    //cudaMallocs
    hipMalloc(&src_data, rows * cols * channels * sizeof(unsigned char));
    hipMalloc(&min_mat_data, rows * cols * sizeof(unsigned char));
    hipMalloc(&dark_channel_mat_data, rows * cols * sizeof(unsigned char));


    //cudaMemcpys
    hipMemcpy(src_data, src_ptr, rows * cols * channels * sizeof(unsigned char), hipMemcpyHostToDevice);



    //Thread and grids decleartion 
    dim3 threadsPerBlock(32, 32);
    dim3 numBlocks((int)ceil((cols + threadsPerBlock.x - 1) / threadsPerBlock.x), (int)ceil((rows + threadsPerBlock.y - 1) / threadsPerBlock.y));



    // kernels of dark channel
    min_channel_kernel<<<numBlocks, threadsPerBlock>>>(src_data, min_mat_data, rows, cols, channels);
    hipDeviceSynchronize();


    hipMalloc(&min_mat_expansion_data, (rows + 2 * border) * (cols + 2 * border) * sizeof(unsigned char));

    padImage<<<numBlocks, threadsPerBlock>>>(min_mat_data, min_mat_expansion_data, rows, cols, border);
    hipDeviceSynchronize();

    min_filter_kernel<<<numBlocks, threadsPerBlock>>>(min_mat_expansion_data, dark_channel_mat_data, rows + 2 * border, cols + 2 * border, border);
    hipDeviceSynchronize();



    hipFree(min_mat_data);
    hipFree(min_mat_expansion_data);


    // kernels of A estimation
    hipMalloc(&sortedIndices, rows * cols * sizeof(int));
    sortIndicesByDarkChannel<<<numBlocks, threadsPerBlock>>>(dark_channel_mat_data, sortedIndices, rows, cols);
    hipDeviceSynchronize();

    

    hipMallocManaged(&A, sizeof(float));
    
    *A = 0.00f;
    findMaxValue<<<numBlocks, threadsPerBlock>>>(src_data, sortedIndices, rows, cols, A);
    hipDeviceSynchronize();

    hipFree(sortedIndices);



    // kernels of tx map
    hipMalloc((void**)&dev_tx_data, rows * cols * channels * sizeof(unsigned char));

    calculate_tx_kernel<<<numBlocks, threadsPerBlock>>>( rows, cols, channels, *A, dark_channel_mat_data, dev_tx_data);
    hipDeviceSynchronize();




    // Kernels for Dehaze formula
    hipMalloc((void**)&result, rows * cols * channels * sizeof(unsigned char));

    
    haze_removal_kernel<<<numBlocks, threadsPerBlock>>>(src_data, rows, cols, channels, *A, dev_tx_data, result);
    
    hipHostAlloc((void **)&result_img, rows * cols * channels * sizeof(unsigned char), hipHostMallocMapped);
    
    hipMemcpy(result_img, result, rows * cols * channels * sizeof(unsigned char), hipMemcpyDeviceToHost);


    hipFree(src_data);
    hipFree(dark_channel_mat_data);
    hipFree(A);
    hipFree(dev_tx_data);
    hipFree(result);


    float milliseconds = measure_kernel_execution_time(start, stop);
    std::cout << "Kernel execution time: " << milliseconds << " ms" << std::endl;

    // Release CUDA events
    hipEventDestroy(start);
    hipEventDestroy(stop);


    return result_img;
}



int main(int argc, char *argv[])
{
	FILE *fp_in, *fp_out;
	
	unsigned char * ori;
	
	hipHostAlloc((void **)&ori, sizeof(unsigned char)*450*600*3, hipHostMallocMapped);
	

	auto fstart = std::chrono::steady_clock::now();
	fp_in = fopen("sample.bmp", "rb");
	fread(ori, 450*600*3, 1, fp_in);
	fclose(fp_in);
	

	
	unsigned char *dehaze = Dehaze_CUDA(ori, 450, 600, 3, 3);
	



	fp_out = fopen("output.bmp", "wb");
	fwrite(dehaze, 450*600*3, 1, fp_out);
	fclose(fp_out);
	
	auto fend = std::chrono::steady_clock::now();

        // Calculate the duration of the function execution
        auto fduration = std::chrono::duration_cast<std::chrono::milliseconds>(fend - fstart);

        // Print the duration
        std::cout << "Function execution duration: " << fduration.count() << " milliseconds" << std::endl;
}
