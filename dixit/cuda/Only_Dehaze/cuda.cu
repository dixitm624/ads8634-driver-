#include<stdio.h>
#include<hip/hip_runtime.h>
#include<hip/hip_runtime.h>

#include<iostream>
#include<ctime>
#include<chrono>

unsigned char* Dehaze_CUDA(unsigned char *src_ptr,int rows, int cols, int channnels, int border)
{
    hipEvent_t
}

int main(int argc, char *argv[]){
    FILE *fp_in , *fp_out;

    unsigned char * ori;

    hipHostAlloc((void **)&ori, sizeof(unsigned char)*40*600*3, hipHostMallocMapped);
    
    auto fstart = std::chrono::steady_clock::now();
    fp_in = fopen("Sample.bmp", "rb");
    fread(ori, 450*600*3, 1, fp_in);
    fclose(fp_in);

    unsigned char *dehaze = Dehaze_CUDA(ori, 450, 600, 3, 3);

    fp_out = fopen("output.bmp", "wb");
    fwrite(dehaze, 450*600*3, 1, fp_out);
    fclose(fp_out);

    auto fend = std::chrono::steady_clock::now();

        auto fduration = std::chrono::duration_cast<std::chrono::milliseconds>(fend -  fstart);

        std::cout << "Function Excetion duration: " << fduration.count() << " milliseconds" << std::endl;

}

